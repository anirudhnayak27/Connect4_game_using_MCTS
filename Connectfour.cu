#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <cmath>
#include <unordered_map>
#include <limits>
#include <random>
#include <chrono>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>


class GameMeta {
public:
    static const std::unordered_map<std::string, int> PLAYERS;
    static const std::unordered_map<std::string, int> OUTCOMES;
    static constexpr double INF = std::numeric_limits<double>::infinity();
    static const int ROWS = 6;
    static const int COLS = 7;
};

const std::unordered_map<std::string, int> GameMeta::PLAYERS = {
    {"none", 0}, {"one", 1}, {"two", 2}
};

const std::unordered_map<std::string, int> GameMeta::OUTCOMES = {
    {"none", 0}, {"one", 1}, {"two", 2}, {"draw", 3}
};

class MCTSMeta {
public:
    static const double EXPLORATION;
};

const double MCTSMeta::EXPLORATION = std::sqrt(2.0);

class ConnectState {
public:
    ConnectState() {
        board = std::vector<std::vector<int>>(GameMeta::ROWS, std::vector<int>(GameMeta::COLS, 0));
        to_play = GameMeta::PLAYERS.at("one");
        height = std::vector<int>(GameMeta::COLS, GameMeta::ROWS - 1);
    }

    std::vector<std::vector<int>> get_board() const {
        return board;
    }

    void move(int col) {
        board[height[col]][col] = to_play;
        last_played = {height[col], col};
        height[col]--;
        to_play = (to_play == GameMeta::PLAYERS.at("one")) ? GameMeta::PLAYERS.at("two") : GameMeta::PLAYERS.at("one");
    }

    std::vector<int> get_legal_moves() const {
        std::vector<int> legal_moves;
        for (int col = 0; col < GameMeta::COLS; ++col) {
            if (board[0][col] == 0) {
                legal_moves.push_back(col);
            }
        }
        return legal_moves;
    }

    int check_win() {
        if (!last_played.empty() && check_win_from(last_played[0], last_played[1])) {
            return board[last_played[0]][last_played[1]];
        }
        return 0;
    }

    bool check_win_from(int row, int col) {
        int player = board[row][col];

        // Check horizontal
        int consecutive = 1;
        int tmprow = row;
        while (tmprow + 1 < GameMeta::ROWS && board[tmprow + 1][col] == player) {
            consecutive++;
            tmprow++;
        }
        tmprow = row;
        while (tmprow - 1 >= 0 && board[tmprow - 1][col] == player) {
            consecutive++;
            tmprow--;
        }
        if (consecutive >= 4) {
            return true;
        }

        // Check vertical
        consecutive = 1;
        int tmpcol = col;
        while (tmpcol + 1 < GameMeta::COLS && board[row][tmpcol + 1] == player) {
            consecutive++;
            tmpcol++;
        }
        tmpcol = col;
        while (tmpcol - 1 >= 0 && board[row][tmpcol - 1] == player) {
            consecutive++;
            tmpcol--;
        }
        if (consecutive >= 4) {
            return true;
        }

        // Check diagonal
        consecutive = 1;
        tmprow = row;
        tmpcol = col;
        while (tmprow + 1 < GameMeta::ROWS && tmpcol + 1 < GameMeta::COLS && board[tmprow + 1][tmpcol + 1] == player) {
            consecutive++;
            tmprow++;
            tmpcol++;
        }
        tmprow = row;
        tmpcol = col;
        while (tmprow - 1 >= 0 && tmpcol - 1 >= 0 && board[tmprow - 1][tmpcol - 1] == player) {
            consecutive++;
            tmprow--;
            tmpcol--;
        }
        if (consecutive >= 4) {
            return true;
        }

        // Check anti-diagonal
        consecutive = 1;
        tmprow = row;
        tmpcol = col;
        while (tmprow + 1 < GameMeta::ROWS && tmpcol - 1 >= 0 && board[tmprow + 1][tmpcol - 1] == player) {
            consecutive++;
            tmprow++;
            tmpcol--;
        }
        tmprow = row;
        tmpcol = col;
        while (tmprow - 1 >= 0 && tmpcol + 1 < GameMeta::COLS && board[tmprow - 1][tmpcol + 1] == player) {
            consecutive++;
            tmprow--;
            tmpcol++;
        }
        if (consecutive >= 4) {
            return true;
        }

        return false;
    }

    bool game_over()  {
        return check_win() != 0 || get_legal_moves().empty();
    }

    int get_outcome()  {
        if (get_legal_moves().empty() && check_win() == 0) {
            return GameMeta::OUTCOMES.at("draw");
        }
        return (check_win() == GameMeta::PLAYERS.at("one")) ? GameMeta::OUTCOMES.at("one") : GameMeta::OUTCOMES.at("two");
    }

    void print() const {
        std::cout << "=============================\n";
        for (int row = 0; row < GameMeta::ROWS; ++row) {
            for (int col = 0; col < GameMeta::COLS; ++col) {
                std::cout << "| " << ((board[row][col] == 1) ? 'X' : (board[row][col] == 2) ? 'O' : ' ') << " ";
            }
            std::cout << "|\n";
        }
        std::cout << "=============================\n";
    }

private:
    std::vector<std::vector<int>> board;
    int to_play;
    std::vector<int> height;
    std::vector<int> last_played;
};

class Node {
public:
    Node(int move, Node* parent)
        : move(move), parent(parent), N(0), Q(0), outcome(GameMeta::PLAYERS.at("none")) {}

    void add_children(const std::vector<Node*>& children) {
        for (Node* child : children) {
            this->children[child->move] = child;
        }
    }

    double value(double explore = MCTSMeta::EXPLORATION) const {
        if (N == 0) {
            return (explore == 0) ? 0 : GameMeta::INF;
        } else {
            return Q / static_cast<double>(N) + explore * std::sqrt(std::log(static_cast<double>(parent->N)) / static_cast<double>(N));
        }
    }

    int move;
    Node* parent;
    int N;
    double Q;
    std::unordered_map<int, Node*> children;
    int outcome;
};



// CUDA error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA kernel to find the index of the maximum value (same as before)
__global__ void find_max_index_kernel(double* values, int num_values, int* max_idx) {
    extern __shared__ char shared_mem[];
    double* shared_max = reinterpret_cast<double*>(shared_mem);
    int* shared_idx = reinterpret_cast<int*>(shared_max + blockDim.x);

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < num_values) {
        shared_max[tid] = values[gid];
        shared_idx[tid] = gid;
    } else {
        shared_max[tid] = -GameMeta::INF;
        shared_idx[tid] = -1;
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && gid < num_values) {
            if (shared_max[tid] < shared_max[tid + s]) {
                shared_max[tid] = shared_max[tid + s];
                shared_idx[tid] = shared_idx[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        *max_idx = shared_idx[0];
    }
}

/*__global__ void find_max_index_kernel(double* values, int num_values, int* max_idx) {
    *max_idx = 0;  // Just return the first index for now
}*/

// Modified function to find max UCT value using CUDA
int find_max_uct_child(const std::unordered_map<int, Node*>& children, Node* parent, double explore = MCTSMeta::EXPLORATION) {
    int num_children = children.size();
    if (num_children == 0) return -1;

    std::vector<int> moves;
    std::vector<double> uct_values;

    for (const auto& pair : children) {
        Node* child = pair.second;
        moves.push_back(pair.first);

        double uct_value;
        if (child == nullptr) {
            uct_value = -GameMeta::INF;
        } else if (child->N == 0) {
            uct_value = (explore == 0) ? 0 : GameMeta::INF;
        } else {
            uct_value = child->Q / static_cast<double>(child->N) +
                       explore * std::sqrt(std::log(static_cast<double>(parent->N)) / static_cast<double>(child->N));
        }
        uct_values.push_back(uct_value);
    }

    // Use managed memory for uct_values
    double* d_uct_values;
    cudaCheckError(hipMallocManaged(&d_uct_values, num_children * sizeof(double)));

    // Copy host data to managed memory
    for (int i = 0; i < num_children; ++i) {
        d_uct_values[i] = uct_values[i];
    }

    // Find max index using our custom kernel
    int* d_max_idx;
    cudaCheckError(hipMallocManaged(&d_max_idx, sizeof(int)));
    *d_max_idx = -1;  // Initialize on host

    int num_threads = 128;
    size_t shared_mem_size = num_threads * (sizeof(double) + sizeof(int));

    find_max_index_kernel<<<1, num_threads, shared_mem_size>>>(
        d_uct_values,
        num_children,
        d_max_idx
    );
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());

    int max_idx = *d_max_idx;  // Copy from managed memory

    // Free managed memory
    cudaCheckError(hipFree(d_uct_values));
    cudaCheckError(hipFree(d_max_idx));

    return moves[max_idx];
}






class MCTS {
public:
    MCTS(const ConnectState& state) : root_state(state) {
        root = new Node(-1, nullptr);
    }

    ~MCTS() {
        deleteNodes(root);
    }

    void set_max_rollouts(int max_rollouts) {
        this->max_rollouts = max_rollouts;
    }


    // Modified select_node function (same as before)
    std::pair<Node*, ConnectState> select_node() {
        Node* node = root;
        ConnectState state = root_state;
        int depth = 0;

        while (!node->children.empty() && depth < GameMeta::ROWS * GameMeta::COLS) {
           // std::cout << "Selecting at depth " << depth << ", children: " << node->children.size() << "\n";
            int max_move = find_max_uct_child(node->children, node);
            //std::cout << "Selected move: " << max_move << "\n";
            node = node->children[max_move];
            state.move(node->move);
            depth++;

            if (node->N == 0) {
               // std::cout << "Returning unexplored node at depth " << depth << "\n";
                return std::make_pair(node, state);
            }
        }

        if (expand(node, state)) {
            node = node->children.begin()->second;
            state.move(node->move);
        }

        return std::make_pair(node, state);
    }




    bool expand(Node* parent, ConnectState& state) {
        if (state.game_over()) {
            return false;
        }

        std::vector<Node*> children;
        for (int move : state.get_legal_moves()) {
          //std::cout << "Adding child with move: " << move << "\n";
          children.push_back(new Node(move, parent));
        }
        parent->add_children(children);

        return true;
    }

    int roll_out(const ConnectState& state) {
        ConnectState rollout_state = state;
        while (!rollout_state.game_over()) {
            std::vector<int> legal_moves = rollout_state.get_legal_moves();
            rollout_state.move(legal_moves[rand() % legal_moves.size()]);
        }
        return rollout_state.get_outcome();
    }

    void back_propagate(Node* node, int turn, int outcome) {
        int reward = (outcome == turn) ? 0 : 1;
        while (node != nullptr) {
            /*std::cout << "Updating node (move=" << node->move << "): N=" << node->N << " -> " << node->N+1
                      << ", Q=" << node->Q << " -> " << node->Q+reward << "\n";*/
            node->N++;
            node->Q += reward;
            node = node->parent;
            if (outcome == GameMeta::OUTCOMES.at("draw")) {
                reward = 0;
            } else {
                reward = 1 - reward;
            }
        }
    }

    void search() {
        auto start_time = std::chrono::steady_clock::now();
        num_rollouts = 0;
        while (!root_state.game_over() && num_rollouts < max_rollouts) {
            //std::cout << "Starting rollout " << num_rollouts << "\n";
            Node* node;
            ConnectState state;
            std::tie(node, state) = select_node();
            int outcome = roll_out(state);
            //std::cout << "Rollout outcome: " << outcome << "\n";
            back_propagate(node, state.get_legal_moves().empty() ? GameMeta::PLAYERS.at("none") : state.get_legal_moves()[0], outcome);
            num_rollouts++;
            if (num_rollouts >= max_rollouts) break;
        }

        run_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start_time).count();
    }

    int best_move()  {

        if (root_state.game_over()) {
            return -1;
        }

        int max_visits = std::max_element(root->children.begin(), root->children.end(), [](const std::pair<int, Node*>& a, const std::pair<int, Node*>& b) {
            return a.second->N < b.second->N;
        })->second->N;

        std::vector<Node*> max_nodes;
        for (const auto& child : root->children) {
            if (child.second->N == max_visits) {
                max_nodes.push_back(child.second);
            }
        }

        return max_nodes[rand() % max_nodes.size()]->move;
    }

    void move(int move) {
        if (root->children.count(move)) {
            root_state.move(move);
            root = root->children[move];
            return;
        }

        root_state.move(move);
        deleteNodes(root);
        root = new Node(-1, nullptr);
    }

    std::pair<int, int> statistics() const {
        return std::make_pair(num_rollouts, run_time);
    }

private:
    void deleteNodes(Node* node) {
        for (const auto& child : node->children) {
            deleteNodes(child.second);
        }
        delete node;
    }

    ConnectState root_state;
    Node* root;
    int num_rollouts;
    int run_time;
    int max_rollouts = 1;
};

void play() {
    ConnectState state;
    MCTS mcts(state);
    mcts.set_max_rollouts(1000);
    std::string quit = "quit";

    while (!state.game_over()) {
        std::cout << "Current state:\n";
        state.print();

        std::string user_move;
        std::vector<int> legal_moves = state.get_legal_moves();
        do {
            std::cout << "Enter a move (or 'quit' to end the game): ";
            std::cin >> user_move;
        } while (user_move != quit && std::find(legal_moves.begin(), legal_moves.end(), std::stoi(user_move)) == legal_moves.end());

        if (user_move == quit) {
            std::cout << "Quitting...\n";
            std::cout << "Player two won!\n";
            state.print();
            break;
        }

        state.move(std::stoi(user_move));
        mcts.move(std::stoi(user_move));
        // state.print();

        if (state.game_over()) {
            std::cout << "Player one won!\n";
            state.print();
            break;
        }

        std::cout << "Thinking...\n";

        mcts.search();
        int num_rollouts, run_time;
        std::tie(num_rollouts, run_time) = mcts.statistics();
        std::cout << "Statistics: " << num_rollouts << " rollouts in " << mcts.statistics().second << " milliseconds\n";
        int move = mcts.best_move();

        std::cout << "MCTS chose move: " << move << "\n";

        state.move(move);
        mcts.move(move);

        if (state.game_over()) {
            std::cout << "Player two won!\n";
            state.print();
            break;
        }
    }
}

int main() {
    std::random_device rd;
    std::mt19937 gen(rd());

    play();

    return 0;
}
